#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#ifdef __NVCC__
#include <hip/hip_runtime_api.h>
#endif

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

//TODO: Remove this hack added for DPC++
#ifdef DPCT_COMPATIBILITY_TEMP
#undef DPCT_COMPATIBILITY_TEMP
#endif

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = hipDeviceSynchronize();
  if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
#endif
}

extern "C" {

  void GetDeviceCount(int *count) {
    CudaSafeCall(hipGetDeviceCount(count));
  }

  void GetDevice(int *device) {
    CudaSafeCall(hipGetDevice(device));
  }

  void SetDevice(int device) {
    CudaSafeCall(hipSetDevice(device));
  }

  void ProfilerStart() {
#ifdef __NVCC__
    CudaSafeCall(hipProfilerStart());
#endif
  }

  void ProfilerStop() {
#ifdef __NVCC__
    CudaSafeCall(hipProfilerStop());
#endif
  }

  void DeviceSynchronize() {
    CudaCheckError();
    CudaSafeCall(hipDeviceSynchronize());
  }

  void Malloc(void** devPtr, size_t size) {
    CudaSafeCall(hipMalloc(devPtr, size));
  }

  void MallocPtr(void*** devPtr, size_t size) {
    CudaSafeCall(hipMalloc(devPtr, size));
  }

  void MallocPtrPtr(void**** devPtr, size_t size) {
    CudaSafeCall(hipMalloc(devPtr, size));
  }

  void MallocPitch(void** devPtr, size_t* pitch, size_t width, size_t height) {
    CudaSafeCall(hipMallocPitch(devPtr, pitch, width, height));
  }

  void MallocUnified(void** umemPtr, size_t size) {
    CudaSafeCall(hipMallocManaged(umemPtr, size));
  }

  void PrefetchToDevice(void* umemPtr, size_t start, size_t end, int device) {
    CudaSafeCall(hipMemPrefetchAsync(static_cast<char *>(umemPtr)+start, end-start, device, NULL));
  }

  void Memcpy(void* dst, void* src, size_t count, int kind) {
      switch (kind) {
      case 0:
          CudaSafeCall(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
          break;
      case 1:
          CudaSafeCall(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
          break;
      default:
          printf("Fatal: Wrong Memcpy kind!\n");
          exit(1);
      }
  }

  void Memcpy2D(void* dst, size_t dpitch, void* src, size_t spitch, size_t width, size_t height, int kind) {
      switch (kind) {
      case 0:
          CudaSafeCall(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyHostToDevice));
          break;
      case 1:
          CudaSafeCall(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost));
          break;
      default:
          printf("Fatal: Wrong Memcpy kind!\n");
          exit(1);
      }
  }

  void Free(void* devPtr) {
      CudaSafeCall(hipFree(devPtr));
  }
}
